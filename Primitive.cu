#include "hip/hip_runtime.h"
#include "Primitive.cuh"

//Parent Primitive
Primitive::Primitive(const ray::vec3 &_loc, const ray::vec3 &_rot, const ray::vec3 &_scale): loc(_loc), rot(_rot), scale(_scale)  {};

// ray::vec3 Primitive::Normal(const ray::vec3 &p) const{
//     float dxp,dxn,dyp,dyn,dzp,dzn;
//     SDF(p + ray::vec3(EPSILON,0.0f,0.0f),nullptr,&dxp);
//     SDF(p - ray::vec3(EPSILON,0.0f,0.0f),nullptr,&dxn);
//
//     SDF(p + ray::vec3(0.0f,EPSILON,0.0f),nullptr,&dyp);
//     SDF(p - ray::vec3(0.0f,EPSILON,0.0f),nullptr,&dyn);
//
//     SDF(p + ray::vec3(0.0f,0.0f,EPSILON),nullptr,&dzp);
//     SDF(p - ray::vec3(0.0f,0.0f,EPSILON),nullptr,&dzn);
//
//
//     return ray::normalize(ray::vec3(dxp-dxn, dyp-dyn, dzp-dzn) );
//     }

ray::vec3 Primitive::getLoc() const {
    return loc;
}

ray::vec3 Primitive::getRot() const {
    return rot;
}

ray::vec3 Primitive::getScale() const {
    return scale;
}

ray::vec3 * Primitive::getLocRef() {
    return &loc;
}

ray::vec3 * Primitive::getRotRef() {
    return &rot;
}

ray::vec3 * Primitive::getScaleRef() {
    return &scale;
}

void Primitive::setLoc(const ray::vec3 &loc) {
    this->loc = loc;
}

void Primitive::setRot(const ray::vec3 &rot) {
    this->rot = rot;
}

void Primitive::setScale(const ray::vec3 &scale) {
    this->scale = scale;
}

Primitive::~Primitive() {}

void Primitive::CubeSDF(const ray::vec3 &p, const ray::vec3 &loc, const ray::vec3 &rot, const ray::vec3 &scale,
    size_t *size, float *out) {

    if (size != nullptr) *size = 1;
    ray::vec3 q(abs(p - loc) - scale);
    out[0] = ray::length(ray::max(q, 0.f)) + min( ray::compMax(q) , 0.0);
}

void Primitive::SphereSDF(const ray::vec3 &p, const ray::vec3 &loc, const ray::vec3 &rot, const ray::vec3 &scale,
    const float radius, size_t *size, float *out) {
    if (size != nullptr) *size = 1;
    out[0]=ray::length(p-loc) - radius;
}

void Primitive::MandelbulbSDF(const ray::vec3 &p, const ray::vec3 &loc, const ray::vec3 &rot, const ray::vec3 &scale,
    const unsigned int iterations, const float exponent, size_t *size, float *out) {
    ray::vec3 pnew =
        ray::rotate(ray::rotate(ray::rotate( (p - loc),0,rot.x * (PI/180.f)),1,rot.y * (PI/180.f)),2,rot.z * (PI/180.f));
    ray::vec3 zold(0.f,0.f,0.f);
    ray::vec3 znew(0.f,0.f,0.f);

    float dr = 1.0f;
    for (unsigned int i = 0; i < iterations; i++) {

        if (ray::length(zold) > 8.f) {
            break;
        }
        znew = (zold ^ exponent) + pnew;
        dr = (exponent * powf(length(zold), exponent-1.f) * dr) + 1.f;
        zold = znew;
    }

    if (size!=nullptr)
        *size = 2;
    out[0] = 0.5f * (ray::length(znew) * logf(ray::length(znew)) )/(dr+EPSILON);
    out[1] =  length(zold) - floorf(length(zold));
}

//Sphere SDF
Sphere::Sphere(const ray::vec3 &_loc, const ray::vec3 &_rot, const ray::vec3 &_scale, const float _radius)
    :Primitive(_loc, _rot, _scale), radius(_radius) {}


PrimitiveType Sphere::getType() const {
    return PrimitiveType::SPHERE;
}

float Sphere::getRadius() const {
    return radius;
}

float * Sphere::getRadiusRef() {
    return &radius;
}

void Sphere::setRadius(const float _radius) {
    this->radius = _radius;
}

Sphere::~Sphere() {}

Cube::Cube(const ray::vec3 &_loc, const ray::vec3 &_rot, const ray::vec3 &_scale)
:Primitive(_loc,_rot,_scale) {}




PrimitiveType Cube::getType() const {
    return PrimitiveType::CUBE;
}

Cube::~Cube() {
}

Mandelbulb::Mandelbulb(const ray::vec3 &_loc, const ray::vec3 &_rot, const ray::vec3 &_scale,
    const unsigned int _iterations, const float _exponent)
        :Primitive(_loc,_rot,_scale), iterations(_iterations), exponent(_exponent){}

unsigned int Mandelbulb::getIterations() const {
    return iterations;
}

unsigned int * Mandelbulb::getIterationsRef() {
    return &iterations;
}

void Mandelbulb::setIterations(const unsigned int _iterations) {
    this->iterations = _iterations;
}

PrimitiveType Mandelbulb::getType() const {
    return PrimitiveType::MANDELBROT;
}

float Mandelbulb::getExponent() const {
    return exponent;
}

float * Mandelbulb::getExponentRef() {
    return &exponent;
}

void Mandelbulb::setExponent(const float _exponent) {
    this->exponent = _exponent;
}

Mandelbulb::~Mandelbulb() {}

